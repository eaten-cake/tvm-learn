#include <stdio.h>
#include <hip/hip_runtime.h>

// 错误检查宏
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    // 定义共享内存
    __shared__ int s_a[256];
    __shared__ int s_b[256];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < n) {
        // 将全局内存的数据加载到共享内存
        s_a[local_tid] = a[tid];
        s_b[local_tid] = b[tid];

        // 由于共享内存加载是并行的，需要同步线程，确保所有数据都已加载
        __syncthreads();

        // 在共享内存中进行计算
        int sum = s_a[local_tid] + s_b[local_tid];

        // 将结果写回全局内存
        c[tid] = sum;
    }
}

int main() {
    const int N = 1024;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    // 分配主机内存并初始化
    h_a = (int*)malloc(N * sizeof(int));
    h_b = (int*)malloc(N * sizeof(int));
    h_c = (int*)malloc(N * sizeof(int));
    for(int i=0; i<N; i++) {
        h_a[i] = i + 1;
        h_b[i] = i + 1;
    }

    // 分配设备内存（添加错误检查）
    CHECK(hipMalloc((void**)&d_a, N * sizeof(int)));
    CHECK(hipMalloc((void**)&d_b, N * sizeof(int)));
    CHECK(hipMalloc((void**)&d_c, N * sizeof(int)));

    // 数据传输 Host→Device
    CHECK(hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice));

    // 启动核函数（修正线程配置）
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, N);

    // 检查核函数错误
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());  // 强制同步等待核函数完成

    // 回传结果
    CHECK(hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost));

    // 验证结果
    bool success = true;
    for(int i=0; i<N; i++) {
        if(h_c[i] != (i+1)*2) {
            printf("Error at index %d: %d != %d\n", i, h_c[i], (i+1)*2);
            success = false;
            break;
        }
    }
    if(success) printf("All elements added correctly!\n");

    // 释放资源
    free(h_a);
    free(h_b);
    free(h_c);
    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));

    return 0;
}
