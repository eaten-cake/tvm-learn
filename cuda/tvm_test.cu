
#include <hip/hip_runtime.h>
#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) main_kernel(float* __restrict__ A, float* __restrict__ B);
extern "C" __global__ void __launch_bounds__(128) main_kernel(float* __restrict__ A, float* __restrict__ B) {
  B[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = ((A[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] + A[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 1)]) + A[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) + 2)]);
}